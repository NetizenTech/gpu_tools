
#include <hip/hip_runtime.h>
#include <cuda_stdint.h>

/* return hash of array - GPU|CPU */
__host__ __device__ uint64_t hash0(const uint8_t *s, const uint16_t N)
{
    uint64_t x = 0;

    for (uint16_t i = 0; i < N; i++)
    {
        x ^= s[i];
        x ^= (x >> 29) & 0x5555555555555555ULL;
        x ^= (x << 17) & 0x71D67FFFEDA60000ULL;
        x ^= (x << 37) & 0xFFF7EEE000000000ULL;
        x ^= (x >> 43);
    }

    return x;
}

/* generate array of hashes from 2D array - GPU-global */
__global__ void hash0_kernel(const uint8_t *s_arr, uint64_t *h_arr, const uint32_t N, const uint16_t NN)
{
    const uint32_t i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N)
        h_arr[i] = hash0((uint8_t *)&s_arr[i * NN], NN);
}

/* search h indexes in array - GPU-global */
__global__ void search_kernel(const uint64_t *h_arr, const uint64_t h, uint32_t *r_arr, const uint32_t N,
                              const uint32_t NN)
{
    const uint32_t i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N)
        if (h_arr[i] == h)
        {
            const uint32_t idx = atomicAdd(&r_arr[0], 1) + 1;
            if (idx < NN)
                r_arr[idx] = i;
        }
}
